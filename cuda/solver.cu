
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>

#define Tolerance 0.00001
#define TRUE 1
#define FALSE 0

#define N 5000

double ** A;

void initialize (double **A, int n)
{
	int i,j;

	for (j=0;j<n+1;j++){
		A[0][j]=1.0;
	}
	for (i=1;i<n+1;i++){
		A[i][0]=1.0;
		for (j=1;j<n+1;j++) A[i][j]=0.0;
	}

}

void solve(double **A, int n)
{
	int convergence=FALSE;
	double diff, tmp;
	int i,j, iters=0;
	int for_iters;


	for (for_iters=1;for_iters<21;for_iters++) 
	{ 
		diff = 0.0;

		for (i=1;i<n;i++)
		{
			for (j=1;j<n;j++)
			{
				tmp = A[i][j];
				A[i][j] = 0.2*(A[i][j] + A[i][j-1] + A[i-1][j] + A[i][j+1] + A[i+1][j]);
				diff += fabs(A[i][j] - tmp);
			}
		}
		iters++;

		if (diff/((double)N*(double)N) < Tolerance)
			convergence=TRUE;

	} /*for*/
}


long usecs (void)
{
	struct timeval t;

	gettimeofday(&t,NULL);
	return t.tv_sec*1000000+t.tv_usec;
}

__global__ void calc_gpu(int* d_arr) {
	d_arr[threadIdx.x]++;
}

int main(int argc, char * argv[])
{
	int i;
	long t_start,t_end;
	double time;

	dim3 grid(10);
	dim3 block(1);
	
	int h_arr[] = {1,2,3,4,5,6,7,8,9,10};
	int *d_arr;
	hipMalloc(&d_arr, 10*sizeof(int));
	hipMemcpy(d_arr, h_arr, 10*sizeof(int), hipMemcpyHostToDevice);
 
	calc_gpu<<<1,10>>>(d_arr);
	
	hipMemcpy(h_arr, d_arr, 10*sizeof(int), hipMemcpyDeviceToHost);
	for (i=0; i < 10; ++i) {
		printf(" %d", h_arr[i]);
	}

	A = (double**)malloc((N+2) * sizeof(double *));
	for (i=0; i<N+2; i++) {
		A[i] = (double*)malloc((N+2) * sizeof(double)); 
	}

	initialize(A, N);

	t_start=usecs();
	solve(A, N);
	t_end=usecs();

	time = ((double)(t_end-t_start))/1000000;
	printf("Computation time = %f\n", time);

}